#include "hip/hip_runtime.h"
#include <iostream>

#include <GL/glew.h>
#include <minmax.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>

#include "MeshResource.h"
#include "Shader.h"
#include "Window.h"
#include "GameTime.h"
#include "DebugCamera.h"

using namespace std;

class VBOTestResource : public MeshResource {
public:
	VBOTestResource();
	~VBOTestResource();

	void draw() const;

	void update();
private:
	void generateGrid();

	unsigned int restart_index;
	int width, height;
	struct hipGraphicsResource* cudaVBO;
};


VBOTestResource::VBOTestResource(): width(1000), height(1000) {

	width = max(2, width);
	height = max(2, height);
	generateGrid();

	cudaGLSetGLDevice(1);

	if (hipGraphicsGLRegisterBuffer(&cudaVBO, vbo[VBO_VERTEX], cudaGraphicsMapFlagsNone) != hipSuccess)
		printf("Failed\n");
}

VBOTestResource::~VBOTestResource() {
	if (hipGraphicsUnregisterResource(cudaVBO) != hipSuccess)
		printf("Failed\n");
}

void VBOTestResource::generateGrid() {

	int loop_size = 2*height + 1;

	int num_verts = width*height;
	int num_indices = (width - 1)*loop_size;

	float* verts = new float[3*num_verts];
	float* norms = new float[3*num_verts];
	float* texcoords = new float[2*num_verts];
	unsigned int* indices = new unsigned int[num_indices];

	mode = GL_TRIANGLE_STRIP;
	for (int x = 0; x < width; x++) {
		int loops = x*loop_size;
		for (int y = 0; y < height; y++) {
			int offset = y*width + x;

			if (x != width - 1)
				indices[loops + 2*y + 1] = offset;
			if (x != 0)
				indices[loops - loop_size + 2*y] = offset;

			verts[3*offset + 0] = 2*(x*1.0f/(width-1)) - 1;
			verts[3*offset + 1] = 0;
			verts[3*offset + 2] = 2*(y*1.0f/(height-1)) - 1;

			norms[3*offset + 0] = 0;
			norms[3*offset + 1] = 1;
			norms[3*offset + 2] = 0;

			texcoords[2*offset + 0] = x*1.0f/(width-1);
			texcoords[2*offset + 1] = y*1.0f/(height-1);
		}
		if (x != width - 1)
			indices[loops + loop_size - 1] = width*height;
	}

	restart_index = width*height;

	glBindVertexArray(vao[0]);
	initVBO(VBO_VERTEX, (float*)verts, 3, num_verts, GL_DYNAMIC_DRAW);
	initVBO(VBO_NORMAL, (float*)norms, 3, num_verts, GL_DYNAMIC_DRAW);
	initVBO(VBO_TEXCOORD, (float*)texcoords, 2, num_verts, GL_DYNAMIC_DRAW);
	initIBO(indices, num_indices, GL_DYNAMIC_DRAW);
	glBindVertexArray(0);

	delete[] verts;
	delete[] norms;
	delete[] texcoords;
	delete[] indices;
}

void VBOTestResource::draw() const {
	glPrimitiveRestartIndex(restart_index);
	glEnable(GL_PRIMITIVE_RESTART);
	MeshResource::draw();
	glDisable(GL_PRIMITIVE_RESTART);
}

__global__ void vboTestResource_update(float* ptr, int width, int height, float time) {
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int offset = y*width + x;
	if (x >= width || y >= height) return;

	float period = 30;
	float rate = 1;

	float cx = x*1.0f/width - 0.5f;
	float cy = y*1.0f/height - 0.5f;

	float wave = sin(sqrt(cx*cx + cy*cy)*period - rate*time);

	int sign = wave>0?1:-1;
	wave = sign*sqrt(sign*wave);

	ptr[3*offset + 1] = wave/10;

	period *= 3;
	rate *= -9;

	ptr[3*offset + 1] += (sin(x*period/(width - 1) + rate*time) + sin(y*period/(height - 1) + rate*time))/40;
}

void VBOTestResource::update() {
	float time = GameTime::getTime();

	float* devBuff;
	size_t size;

	dim3 threadsPerBlock(8, 8);
	dim3 numBlocks((width - 1)/threadsPerBlock.x + 1, (height - 1)/threadsPerBlock.y + 1);

	if (hipGraphicsMapResources(1, &cudaVBO, 0) != hipSuccess)
		printf("Failed\n");

	hipGraphicsResourceGetMappedPointer((void**)&devBuff, &size, cudaVBO);

	vboTestResource_update<<<numBlocks, threadsPerBlock>>>(devBuff, width, height, time);

	if (hipGraphicsUnmapResources(1, &cudaVBO, 0) != hipSuccess)
		printf("Failed\n");
}

const char vert[] =
	"#version 150 core\n"
	"uniform mat4 model;\n"
	"uniform mat4 view;\n"
	"uniform mat4 proj;\n"
	"in vec3 position;\n"
	"in vec3 normal;\n"
	"in vec2 texCoord;\n"
	"out Vertex {\n"
	"	vec3 norm;\n"
	"	vec3 worldPos;\n"
	"	vec2 texCoord;\n"
	"} OUT;\n"
	"void main() {\n"
	"	mat4 mv = view*model;\n"
	"	vec4 worldPos = model*vec4(position, 1);\n"
	"	gl_Position = proj*view*worldPos;\n"
	"	OUT.norm = normalize(mat3(model)*normal);\n"
	"	OUT.texCoord = texCoord;\n"
	"	OUT.worldPos = worldPos.xyz;\n"
	"}\n";
const char frag[] =
	"#version 150 core\n"
	"uniform sampler2D diffTex;\n"
	"in Vertex {\n"
	"	vec3 norm;\n"
	"	vec3 worldPos;\n"
	"	vec2 texCoord;\n"
	"} IN;\n"
	"out vec4 fragColour;\n"
	"void main() {\n"
	"	fragColour = vec4(sin(IN.worldPos.x), cos(IN.worldPos.y), sin(IN.worldPos.z), 1);\n"
	"}\n";

int main() {
	int swidth, sheight;
	Window win(swidth = 1280, sheight = 720);

	GLenum err = glewInit();
	if(err != GLEW_OK) {
		fprintf(stderr, "GLEW Error: %s\n", glewGetErrorString(err));
		return -1;
	}


	Shader shader(vert, frag);
	shader.use();
	DebugCamera dbcam(Vector3(10, 10, 10), 3.14f/4, -7*3.14f/32);

	int modelLoc = glGetUniformLocation(shader.getProgram(), "model");
	int viewLoc = glGetUniformLocation(shader.getProgram(), "view");
	int projLoc = glGetUniformLocation(shader.getProgram(), "proj");
	
	Matrix4 model = Matrix4::diagonal(10,10,10,1);
	Matrix4 proj = Matrix4::perspective(60, sheight*1.0f/swidth, 0.1f, 1000);
	Matrix4 view;
	
	shader.send(modelLoc, 1, &model);
	shader.send(projLoc, 1, &proj);
	
	VBOTestResource test;
	win.show();

	glEnable(GL_DEPTH_TEST);

	GameTime::startTime();

	int running = true;
	while (running) {

		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		
		GameTime::startFrame();

		view = dbcam.getTransformMatrix();
		shader.send(viewLoc, 1, &view);

		dbcam.updateView();
		test.update();
		shader.use();
		test.draw();

		win.swapBuffers();
		win.update();
	}

	return 0;
}